#include "hip/hip_runtime.h"
#include "CUDA/kernel.cuh"

#include <deque>

#include <>
#include <glm/vec2.hpp>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "Node.cuh"
#include "List.cuh"
#include "Map.cuh"
#include "Agent.cuh"
#include "Timer.h"

namespace spark {
	namespace cuda {

		__global__ void checkMapValues(Map* mapDev)
		{
			int index = 0;
			for (int i = 0; i < mapDev->width; ++i)
			{
				for (int j = 0; j < mapDev->height; ++j)
				{
					int resultIndex = mapDev->getTerrainNodeIndex(i, j);
					if (resultIndex == index)
					{
						if (mapDev->nodes[resultIndex] == 1.0f)
						{
							float f = mapDev->getTerrainNodeIndex(i, j);
						}
					}
					++index;
				}
			}
		}

		__global__  void attachNodes(Map* map, float* nodes)
		{
			map->nodes = nodes;
		}

		__global__ void findPath(Map* map, int* path, int* memSize, Agent* agents)
		{
			extern __shared__ int closedNodesLookup[];

			int integers[] = { 8, 5, 0, 2};
			thrust::sort(thrust::seq, integers, integers + 4);


			int startPoint[] = { agents[0].points[0], agents[0].points[1] };
			int endPoint[] = { agents[0].points[2], agents[0].points[3] };

			const Node startNode(startPoint, 0.0f);
			List<Node> openNodes;
			List<Node> closedNodes;
			Node* finishNode = nullptr;
			openNodes.insert(startNode);

			while(true)
			{
				if(openNodes.size == 0)
				{
					break;
				}

				const auto closedNode = openNodes.pop_front();
				closedNodes.insert(closedNode);
				
				unsigned int beforeChange = closedNodesLookup[map->getTerrainNodeIndex(closedNode->value.pos[0], closedNode->value.pos[1])];
				const unsigned int change = beforeChange | (1 << threadIdx.x);
				closedNodesLookup[map->getTerrainNodeIndex(closedNode->value.pos[0], closedNode->value.pos[1])] = beforeChange | change;

				if(closedNode->value.pos[0] == endPoint[0] &&
					closedNode->value.pos[1] == endPoint[1])
				{
					finishNode = &closedNode->value;
					break;
				}

				const auto neighbors = closedNode->value.getNeighbors(map);

				for (auto neighborIt = neighbors.first; neighborIt != nullptr; neighborIt = neighborIt->next)
				{
					/*nvstd::function<bool(const Node& node)> f = [neighborIt] __device__ (const Node& node)
					{
						return node.pos[0] == neighborIt->value.pos[0] &&
							node.pos[1] == neighborIt->value.pos[1];
					};*/
					if (closedNodesLookup[map->getTerrainNodeIndex(neighborIt->value.pos[0], neighborIt->value.pos[1])] & (1 << threadIdx.x))
					{
						continue;
					}

					/*const auto neighborClosed = closedNodes.find(neighborIt->value);
					if (neighborClosed != nullptr)
						continue;*/

					neighborIt->value.parent = &closedNode->value;

					neighborIt->value.valueH = neighborIt->value.measureManhattanDistance(endPoint);
					const float functionG = neighborIt->value.distanceFromBeginning;
					const int neighborPos[2] = { neighborIt->value.pos[0], neighborIt->value.pos[1] };
					const float terrainValue = map->getTerrainValue(neighborPos[0], neighborPos[1]);
					neighborIt->value.valueF = (1.0f - terrainValue) * (neighborIt->value.valueH + functionG);
					

					nvstd::function<bool(const Node& node)> findOpened = [neighborIt] __device__ (const Node& node)
					{
						const bool positionEqual = node.pos[0] == neighborIt->value.pos[0] &&
							node.pos[1] == neighborIt->value.pos[1];
						
						if (!positionEqual)
						{
							return false;
						}

						const bool betterFunctionG = neighborIt->value.distanceFromBeginning < node.distanceFromBeginning;
						
						return betterFunctionG;
					};

					const auto betterNode = openNodes.find_if(findOpened);
					if (betterNode != nullptr)
					{
						openNodes.remove(betterNode);
						openNodes.insert(neighborIt->value);
					}
					else
					{
						openNodes.insert(neighborIt->value);
					}
				}
			}
			int pathLength = 0;
			if(!finishNode)
			{
				return;
			}

			finishNode->getPathLength(pathLength);
			int* tab = new int[2 * pathLength]; //x,y * length
			finishNode->recreatePath(tab, pathLength);
			atomicAdd(memSize, pathLength * 2);

			agents[0].pathOutput = tab;
			agents[0].pathSize = pathLength;
		}

		__global__ void fillPathBuffer(Agent* agents, int* pathBuffer, int numberOfAgents)
		{
			int pathIndex = 0;
			for(int i = 0; i < numberOfAgents; ++i)
			{
				agents[i].indexBegin = pathIndex;
				/*for (int j = 0; j < agents[i].pathSize * 2; ++j)
				{
					pathBuffer[pathIndex] = agents[i].pathOutput[j];
					++pathIndex;
				}*/
				memcpy(pathBuffer + agents[i].indexBegin, agents[i].pathOutput, agents[i].pathSize * 2);
				pathIndex += agents[i].pathSize * 2;
				delete[] agents[i].pathOutput;
			}
		}

		void runKernel(Map* map, float* nodes, int* path, int* memSize, Agent* agents)
		{
			{
				Timer t3("CUDA KERNEL: attach nodes and findPath");
				attachNodes <<<1, 1 >>> (map, nodes);
				hipDeviceSynchronize();
				//checkMapValues <<<1, 1 >>> (map);
				findPath <<<1, 1, 400 * sizeof(unsigned int)>>> (map, path, memSize, agents);
				hipDeviceSynchronize();
			}
			int memorySize = 0;
			int* pathBuffer = nullptr;
			{
				Timer t("CUDA KERNEL: filling pathBuffer");
				
				hipMemcpy(&memorySize, memSize, sizeof(int), hipMemcpyDeviceToHost);
				
				hipMalloc(&pathBuffer, sizeof(int) * memorySize);
				//hipDeviceSynchronize();
				fillPathBuffer <<<1, 1>>> (agents, pathBuffer, 1);
				hipDeviceSynchronize();
			}
			
			{
				Timer t2("CUDA KERNEL: Filling agent path");
				Agent* agentLookup = new Agent[1];
				hipMemcpy(agentLookup, agents, sizeof(Agent), hipMemcpyDeviceToHost);
				int* paths = new int[memorySize];
				hipMemcpy(paths, pathBuffer, sizeof(int) * memorySize, hipMemcpyDeviceToHost);

				for (int i = 0; i < 1; ++i)
				{
					std::deque<std::pair<bool, glm::ivec2>> agentPath;
					for (int j = agentLookup[i].indexBegin; j < agentLookup[i].pathSize * 2; j += 2)
					{
						agentPath.push_back({ false, {paths[j], paths[j + 1]} });
					}
				}

				delete[] paths;
				hipFree(pathBuffer);
			}
		}
	}
}
