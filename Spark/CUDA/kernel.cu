#include "hip/hip_runtime.h"
#include "CUDA/kernel.cuh"

#include <deque>

#include <>

#include "Agent.cuh"
#include "DeviceTimer.cuh"
#include "List.cuh"
#include "Map.cuh"
#include "MemoryAllocator.cuh"
#include "Node.cuh"
#include "Timer.h"

namespace spark {
	namespace cuda {
		__device__ Map* map = nullptr;

		__host__ void runKernel(int* path, int* memSize, Agent* agents)
		{

			//checkMapValues <<<1, 1 >>> (map);
			hipDeviceSynchronize();
			Timer t3("	findPath");
			findPath << <1, 1, 400 * sizeof(unsigned int) >> > (path, memSize, agents);
			gpuErrchk(hipGetLastError());
			hipDeviceSynchronize();
			//int memorySize = 0;
			//int* pathBuffer = nullptr;
			//{
			//	Timer t("CUDA KERNEL: filling pathBuffer");
			//	
			//	hipMemcpy(&memorySize, memSize, sizeof(int), hipMemcpyDeviceToHost);
			//	
			//	hipMalloc(&pathBuffer, sizeof(int) * memorySize);
			//	//hipDeviceSynchronize();
			//	fillPathBuffer <<<1, 1>>> (agents, pathBuffer, 1);
			//	hipDeviceSynchronize();
			//}
			//
			//{
			//	Timer t2("CUDA KERNEL: Filling agent path");
			//	Agent* agentLookup = new Agent[1];
			//	hipMemcpy(agentLookup, agents, sizeof(Agent), hipMemcpyDeviceToHost);
			//	int* paths = new int[memorySize];
			//	hipMemcpy(paths, pathBuffer, sizeof(int) * memorySize, hipMemcpyDeviceToHost);

			//	for (int i = 0; i < 1; ++i)
			//	{
			//		std::deque<std::pair<bool, glm::ivec2>> agentPath;
			//		for (int j = agentLookup[i].indexBegin; j < agentLookup[i].pathSize * 2; j += 2)
			//		{
			//			agentPath.push_back({ false, {paths[j], paths[j + 1]} });
			//		}
			//	}

			//	delete[] paths;
			//	hipFree(pathBuffer);
		}

		__host__ void initMap(float* nodes, int width, int height)
		{
			Timer t("		initMap");
			createMap << <1, 1 >> > (nodes, width, height);
			gpuErrchk(hipGetLastError());
		}

		__global__  void createMap(float* nodes, int width, int height)
		{
			if (map != nullptr)
			{
				delete map;
			}
			map = new Map();
			map->nodes = new float[width * height];
			memcpy(map->nodes, nodes, width * height * sizeof(float));
			//map->nodes = nodes;
			map->width = width;
			map->height = height;
		}

		__global__ void findPath(int* path, int* memSize, Agent* agents)
		{
			extern __shared__ int closedNodesLookup[];
			/*int startPoint[] = { *(path + 4 * threadIdx.x + 0), *(path + 4 * threadIdx.x + 1) };
			int endPoint[] = { *(path + 4 * threadIdx.x + 2), *(path + 4* threadIdx.x + 3) };*/
			DeviceTimer timer2;
			int startPoint[] = { *(path + 0), *(path + 1) };
			int endPoint[] = { *(path + 2), *(path + 3) };
			DeviceTimer timer;
			MemoryAllocator allocator = MemoryAllocator(sizeof(Node) * 400 * 8);
			timer.printTime("Memory allocation %f ms\n");

			const Node startNode(startPoint, 0.0f);
			List<Node> openNodes(&allocator);
			List<Node> closedNodes(&allocator);
			Node* finishNode = nullptr;
			openNodes.insert(startNode);

			while (true)
			{
				DeviceTimer loopTimer;
				if (openNodes.size == 0)
				{
					break;
				}

				const auto closedNode = openNodes.pop_front();
				closedNodes.insert(closedNode);

				//timer.reset();
				unsigned int beforeChange = closedNodesLookup[map->getTerrainNodeIndex(closedNode->value.pos[0], closedNode->value.pos[1])];
				const unsigned int change = beforeChange | (1 << threadIdx.x);
				closedNodesLookup[map->getTerrainNodeIndex(closedNode->value.pos[0], closedNode->value.pos[1])] = beforeChange | change;
				//timer.printTime("Bitwise setting that node is now closed %f ms\n");

				if (closedNode->value.pos[0] == endPoint[0] &&
					closedNode->value.pos[1] == endPoint[1])
				{
					finishNode = &closedNode->value;
					break;
				}

				timer.reset();
				Node neighbors[8];
				closedNode->value.getNeighbors(map, neighbors);
				timer.printTime("Getting node neighbors %f ms\n");

				for (int i = 0; i < 8; ++i)
				{
					/*nvstd::function<bool(const Node& node)> f = [neighborIt] __device__ (const Node& node)
					{
						return node.pos[0] == neighborIt->value.pos[0] &&
							node.pos[1] == neighborIt->value.pos[1];
					};*/
					if (!neighbors[i].valid)
					{
						continue;
					}

					if (closedNodesLookup[map->getTerrainNodeIndex(neighbors[i].pos[0], neighbors[i].pos[1])] & (1 << threadIdx.x))
					{
						continue;
					}

					timer.reset();
					neighbors[i].parent = &closedNode->value;

					neighbors[i].valueH = neighbors[i].measureManhattanDistance(endPoint);
					const float functionG = neighbors[i].distanceFromBeginning;
					const int neighborPos[2] = { neighbors[i].pos[0], neighbors[i].pos[1] };
					const float terrainValue = map->getTerrainValue(neighborPos[0], neighborPos[1]);
					neighbors[i].valueF = (1.0f - terrainValue) * (neighbors[i].valueH + functionG);
					timer.printTime("	Heuristic calculation %f ms\n");

					nvstd::function<bool(const Node& node)> findOpened = [&neighbors, i] __device__(const Node& node)
					{
						const bool positionEqual = node.pos[0] == neighbors[i].pos[0] &&
							node.pos[1] == neighbors[i].pos[1];

						if (!positionEqual)
						{
							return false;
						}

						const bool betterFunctionG = neighbors[i].distanceFromBeginning < node.distanceFromBeginning;

						return betterFunctionG;
					};
					
					timer.reset();
					const auto betterNode = openNodes.find_if(findOpened);
					timer.printTime("	Find better opened node + lambda %f ms\n");

					if (betterNode != nullptr)
					{
						timer.reset();
						openNodes.remove(betterNode);
						openNodes.insert(neighbors[i]);
						timer.printTime("	Node insertion after node deletion %f ms\n");
					}
					else
					{	
						timer.reset();
						openNodes.insert(neighbors[i]);
						timer.printTime("	Node insertion %f ms\n");
					}
				}
				loopTimer.printTime("While loop iteration %f ms\n");
			}

			if (!finishNode)
			{
				return;
			}

			const int pathLength = finishNode->distanceFromBeginning + 1;
			timer2.printTime("Kernel overall time %f ms\n");
			//printf("path length %d\nopened nodes %d\nclosed nodes %d\n", pathLength, openNodes.size, closedNodes.size);
			
			//finishNode->getPathLength(pathLength);
			//int* tab = new int[2 * pathLength]; //x,y * length
			//finishNode->recreatePath(tab, pathLength);
			//atomicAdd(memSize, pathLength * 2);

			//agents[0].pathOutput = tab;
			//agents[0].pathSize = pathLength;
		}

		__global__ void fillPathBuffer(Agent* agents, int* pathBuffer, int numberOfAgents)
		{
			int pathIndex = 0;
			for (int i = 0; i < numberOfAgents; ++i)
			{
				agents[i].indexBegin = pathIndex;
				/*for (int j = 0; j < agents[i].pathSize * 2; ++j)
				{
					pathBuffer[pathIndex] = agents[i].pathOutput[j];
					++pathIndex;
				}*/
				memcpy(pathBuffer + agents[i].indexBegin, agents[i].pathOutput, agents[i].pathSize * 2);
				pathIndex += agents[i].pathSize * 2;
				delete[] agents[i].pathOutput;
			}
		}

		__global__ void checkMapValues(Map* mapDev)
		{
			int index = 0;
			for (int i = 0; i < mapDev->width; ++i)
			{
				for (int j = 0; j < mapDev->height; ++j)
				{
					int resultIndex = mapDev->getTerrainNodeIndex(i, j);
					if (resultIndex == index)
					{
						if (mapDev->nodes[resultIndex] == 1.0f)
						{
							float f = mapDev->getTerrainNodeIndex(i, j);
						}
					}
					++index;
				}
			}
		}
	}
}

